# include <hip/hip_runtime.h>
# include <stdio.h>


__global__ void addInt(int *a, int *b)
{
    a[0] += b[0];
}

int main(int argc, char **argv)
{
    int a = 5;
    int b = 9;

    int *d_a, *d_b;

    hipMalloc(&d_a,sizeof(int));
    hipMalloc(&d_b,sizeof(int));

    hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);

    addInt<<<1,1>>>(d_a,d_b);

    hipMemcpy(&a,d_a,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    printf("The awnser is %i\n",a);

    return 0;
}