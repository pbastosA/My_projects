# include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>


__global__ void addInts(int * a, int * b, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index < n) a[index] += b[index];
}

int main(int argc, char **argv)
{
    int n = 100;

    int * h_a = (int *) malloc(n*sizeof(int)); 
    int * h_b = (int *) malloc(n*sizeof(int)); 

    for (int i = 0; i < n; i++)
    {
        h_a[i] = rand() % 1000;
        h_b[i] = rand() % 1000;
    }

    int * d_a;
    int * d_b;

    hipMalloc(&d_a,n*sizeof(int));
    hipMalloc(&d_b,n*sizeof(int));
    
    hipMemcpy(d_a,h_a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,n*sizeof(int),hipMemcpyHostToDevice);

    addInts<<<1,n>>>(d_a,d_b,n);

    hipMemcpy(h_a,d_a,n*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(h_b,d_b,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("Results of summation:\n");
    for (int i = n-10; i < n; i++) 
        printf("%d + %d = %d\n",h_a[i]-h_b[i],h_b[i],h_a[i]);

    delete[] h_a;
    delete[] h_b;

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}