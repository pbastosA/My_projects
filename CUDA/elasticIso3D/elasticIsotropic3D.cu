#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include <time.h>
# include <math.h>
# include <stdio.h>
# include <stdlib.h>
# include "auxiliaries/functions.h"

int main(int argc,char**argv) 
{
    float total_time;
    time_t t_0, t_f;
    t_0 = time(NULL);
    
    int nx,ny,nz,nt,nabc,wbh;
    int nrecx,nrecy,nsrc;
    float dx,dy,dz,dt;

    readParameters(&nx,&ny,&nz,&nt,&dx,&dy,&dz,&dt,&nabc,&nrecx,&nrecy,&nsrc,&wbh,argv[1]);

    int nxx = nx + 2*nabc; 
    int nyy = ny + 2*nabc; 
    int nzz = nz + 2*nabc;

    int nrecs = nrecx*nrecy;
    int nshot = (nrecx-1)*(nrecy-1);

    /* Host arrays */

    float *vp   = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *vs   = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *rho  = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *M    = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *L    = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *damp = (float *) malloc(nxx*nyy*nzz*sizeof(float));

    float *Vx  = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Vy  = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Vz  = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Txx = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Tyy = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Tzz = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Txz = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Tyz = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Txy = (float *) malloc(nxx*nyy*nzz*sizeof(float));

    float *P   = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Sv  = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Shx = (float *) malloc(nxx*nyy*nzz*sizeof(float));
    float *Shy = (float *) malloc(nxx*nyy*nzz*sizeof(float)); 
    float *Ps  = (float *) malloc(nxx*nyy*nzz*sizeof(float));

    float *source = (float *) malloc(nsrc*sizeof(float));
    
    int *xsrc = (int *) malloc(nshot*sizeof(int)); 
    int *ysrc = (int *) malloc(nshot*sizeof(int)); 
    int *zsrc = (int *) malloc(nshot*sizeof(int));
    int *xrec = (int *) malloc(nrecs*sizeof(int)); 
    int *yrec = (int *) malloc(nrecs*sizeof(int)); 
    int *zrec = (int *) malloc(nrecs*sizeof(int)); 

    float *seismPs = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVx = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVy = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVz = (float *) malloc(nt*nrecs*sizeof(float));
    
    float *seismP   = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismSv  = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismShx = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismShy = (float *) malloc(nt*nrecs*sizeof(float));

    /* Device arrays */

    float *d_Vx;  hipMalloc(&d_Vx,nxx*nyy*nzz*sizeof(float));
    float *d_Vy;  hipMalloc(&d_Vy,nxx*nyy*nzz*sizeof(float));
    float *d_Vz;  hipMalloc(&d_Vz,nxx*nyy*nzz*sizeof(float));
    float *d_Txx; hipMalloc(&d_Txx,nxx*nyy*nzz*sizeof(float));
    float *d_Tyy; hipMalloc(&d_Tyy,nxx*nyy*nzz*sizeof(float));
    float *d_Tzz; hipMalloc(&d_Tzz,nxx*nyy*nzz*sizeof(float));
    float *d_Txz; hipMalloc(&d_Txz,nxx*nyy*nzz*sizeof(float));
    float *d_Tyz; hipMalloc(&d_Tyz,nxx*nyy*nzz*sizeof(float));
    float *d_Txy; hipMalloc(&d_Txy,nxx*nyy*nzz*sizeof(float));

    float *d_P;   hipMalloc(&d_P,nxx*nyy*nzz*sizeof(float));
    float *d_Sv;  hipMalloc(&d_Sv,nxx*nyy*nzz*sizeof(float));
    float *d_Shx; hipMalloc(&d_Shx,nxx*nyy*nzz*sizeof(float));
    float *d_Shy; hipMalloc(&d_Shy,nxx*nyy*nzz*sizeof(float)); 
    float *d_Ps;  hipMalloc(&d_Ps,nxx*nyy*nzz*sizeof(float));

    float *d_rho;  hipMalloc(&d_rho,nxx*nyy*nzz*sizeof(float));
    float *d_M;    hipMalloc(&d_M,nxx*nyy*nzz*sizeof(float)); 
    float *d_L;    hipMalloc(&d_L,nxx*nyy*nzz*sizeof(float)); 
    float *d_damp; hipMalloc(&d_damp,nxx*nyy*nzz*sizeof(float)); 

    float *d_source; hipMalloc(&d_source,nsrc*sizeof(float));
    
    int *d_xsrc; hipMalloc(&d_xsrc,nshot*sizeof(int)); 
    int *d_ysrc; hipMalloc(&d_ysrc,nshot*sizeof(int)); 
    int *d_zsrc; hipMalloc(&d_zsrc,nshot*sizeof(int));
    int *d_xrec; hipMalloc(&d_xrec,nrecs*sizeof(int)); 
    int *d_yrec; hipMalloc(&d_yrec,nrecs*sizeof(int)); 
    int *d_zrec; hipMalloc(&d_zrec,nrecs*sizeof(int)); 

    float *d_seismPs; hipMalloc(&d_seismPs,nt*nrecs*sizeof(float));
    float *d_seismVx; hipMalloc(&d_seismVx,nt*nrecs*sizeof(float));
    float *d_seismVy; hipMalloc(&d_seismVy,nt*nrecs*sizeof(float));
    float *d_seismVz; hipMalloc(&d_seismVz,nt*nrecs*sizeof(float));
    
    float *d_seismP;   hipMalloc(&d_seismP,nt*nrecs*sizeof(float));
    float *d_seismSv;  hipMalloc(&d_seismSv,nt*nrecs*sizeof(float));
    float *d_seismShx; hipMalloc(&d_seismShx,nt*nrecs*sizeof(float));
    float *d_seismShy; hipMalloc(&d_seismShy,nt*nrecs*sizeof(float));








    // importFloatVector(vp,nxx*nyy*nzz,argv[2]);
    // importFloatVector(vs,nxx*nyy*nzz,argv[3]);
    // importFloatVector(rho,nxx*nyy*nzz,argv[4]);
    // importFloatVector(damp,nxx*nyy*nzz,argv[5]);

    // importFloatVector(source,nsrc,argv[6]);
    // importIntegerVector(xsrc,nshot,argv[7]);
    // importIntegerVector(ysrc,nshot,argv[8]);
    // importIntegerVector(xrec,nrecs,argv[9]);
    // importIntegerVector(yrec,nrecs,argv[10]);

    // for (int index = 0; index < nxx*nyy*nzz; index++)
    // {
    //     M[index] = rho[index]*pow(vs[index],2.0f);
    //     L[index] = rho[index]*pow(vp[index],2.0f) - 2.0f*M[index];
    // }

    // for (int i = 0; i < nshot; i++) zsrc[i] = nabc + 5;
    // for (int i = 0; i < nrecs; i++) zrec[i] = nabc + wbh;
    
    /* Managing memory */
    







    // for (int shotPointer = floor(nshot/2 - nrecx/2); shotPointer < floor(nshot/2 - nrecx/2 + 1); shotPointer++)
    // {
    //     setWaveField(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,nxx*nyy*nzz);

    //     for (int timePointer = 0; timePointer < nt; timePointer++) 
    //     {
    //         if (timePointer % (nt/10) == 0) printf("Propagation time = %.3f\n",timePointer*dt);

    //         FDM8E2T_elasticIsotropic3D(Vx,Vy,Vz,Txx,Tyy,Tzz,Txy,Txz,Tyz,rho,M,L,source,xsrc,ysrc,zsrc,nsrc,timePointer,shotPointer,nxx,nyy,nzz,dx,dy,dz,dt);        
    //         cerjanElasticAbsorbingCondition3D(Vx,Vy,Vz,Txx,Tyy,Tzz,Txy,Txz,Tyz,damp,nxx*nyy*nzz);
                
    //         getPressureWaveField(Txx,Tyy,Tzz,Ps,nxx*nyy*nzz);
    //         getPWaveField(Vx,Vy,Vz,P,nxx,nyy,nzz,dx,dy,dz);
    //         getSWaveField(Vx,Vy,Vz,Shx,Shy,Sv,nxx,nyy,nzz,dx,dy,dz);

    //         getSeismogram(seismVx,Vx,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismVy,Vy,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismVz,Vz,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismPs,Ps,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);

    //         getSeismogram(seismP,P,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismSv,Sv,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismShx,Shx,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismShy,Shy,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //     }
    // }
    
    // exportVector(seismVx,nt*nrecs,(char *)"results/seismVx.bin");
    // exportVector(seismVy,nt*nrecs,(char *)"results/seismVy.bin");
    // exportVector(seismVz,nt*nrecs,(char *)"results/seismVz.bin");
    // exportVector(seismPs,nt*nrecs,(char *)"results/seismPs.bin");

    // exportVector(seismP,nt*nrecs,(char *)"results/seismP.bin");
    // exportVector(seismSv,nt*nrecs,(char *)"results/seismSv.bin");
    // exportVector(seismShx,nt*nrecs,(char *)"results/seismShx.bin");
    // exportVector(seismShy,nt*nrecs,(char *)"results/seismShy.bin");

    t_f = time(NULL);
    total_time = difftime(t_f, t_0);
    printf("\nExecution time: \033[31m%.0fs\n\033[m", total_time);    
    return 0;
} 