#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include <time.h>
# include <math.h>
# include <stdio.h>
# include <stdlib.h>
# include <hip/hip_runtime.h>
# include "auxiliaries/functions.h"

int main(int argc,char**argv) 
{
    float total_time;
    time_t t_0, t_f;
    t_0 = time(NULL);
    
    int nx,ny,nz,nt,nabc,wbh;
    int nrecx,nrecy,nsrc;
    float dx,dy,dz,dt;

    readParameters(&nx,&ny,&nz,&nt,&dx,&dy,&dz,&dt,&nabc,&nrecx,&nrecy,&nsrc,&wbh,argv[1]);

    int nxx = nx + 2*nabc; 
    int nyy = ny + 2*nabc; 
    int nzz = nz + 2*nabc;
    
    int nPoints = nxx*nyy*nzz;
    int threads = 1000;
    
    int nrecs = nrecx*nrecy;
    int nshot = (nrecx-1)*(nrecy-1);

    /* Host arrays */

    float *vp   = (float *) malloc(nPoints*sizeof(float));
    float *vs   = (float *) malloc(nPoints*sizeof(float));
    float *rho  = (float *) malloc(nPoints*sizeof(float));
    float *M    = (float *) malloc(nPoints*sizeof(float));
    float *L    = (float *) malloc(nPoints*sizeof(float));
    float *damp = (float *) malloc(nPoints*sizeof(float));

    float *source = (float *) malloc(nsrc*sizeof(float));
    
    int *xsrc = (int *) malloc(nshot*sizeof(int)); 
    int *ysrc = (int *) malloc(nshot*sizeof(int)); 
    int *zsrc = (int *) malloc(nshot*sizeof(int));
    int *xrec = (int *) malloc(nrecs*sizeof(int)); 
    int *yrec = (int *) malloc(nrecs*sizeof(int)); 
    int *zrec = (int *) malloc(nrecs*sizeof(int)); 

    float *seismPs = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVx = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVy = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismVz = (float *) malloc(nt*nrecs*sizeof(float));
    
    float *seismP   = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismSv  = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismShx = (float *) malloc(nt*nrecs*sizeof(float));
    float *seismShy = (float *) malloc(nt*nrecs*sizeof(float));

    /* Device arrays */

    float *d_Vx;  hipMalloc(&d_Vx,nPoints*sizeof(float));
    float *d_Vy;  hipMalloc(&d_Vy,nPoints*sizeof(float));
    float *d_Vz;  hipMalloc(&d_Vz,nPoints*sizeof(float));
    float *d_Txx; hipMalloc(&d_Txx,nPoints*sizeof(float));
    float *d_Tyy; hipMalloc(&d_Tyy,nPoints*sizeof(float));
    float *d_Tzz; hipMalloc(&d_Tzz,nPoints*sizeof(float));
    float *d_Txz; hipMalloc(&d_Txz,nPoints*sizeof(float));
    float *d_Tyz; hipMalloc(&d_Tyz,nPoints*sizeof(float));
    float *d_Txy; hipMalloc(&d_Txy,nPoints*sizeof(float));

    float *d_P;   hipMalloc(&d_P,nPoints*sizeof(float));
    float *d_Sv;  hipMalloc(&d_Sv,nPoints*sizeof(float));
    float *d_Shx; hipMalloc(&d_Shx,nPoints*sizeof(float));
    float *d_Shy; hipMalloc(&d_Shy,nPoints*sizeof(float)); 
    float *d_Ps;  hipMalloc(&d_Ps,nPoints*sizeof(float));

    float *d_rho;  hipMalloc(&d_rho,nPoints*sizeof(float));
    float *d_M;    hipMalloc(&d_M,nPoints*sizeof(float)); 
    float *d_L;    hipMalloc(&d_L,nPoints*sizeof(float)); 
    float *d_damp; hipMalloc(&d_damp,nPoints*sizeof(float)); 

    float *d_source; hipMalloc(&d_source,nsrc*sizeof(float));
    
    int *d_xsrc; hipMalloc(&d_xsrc,nshot*sizeof(int)); 
    int *d_ysrc; hipMalloc(&d_ysrc,nshot*sizeof(int)); 
    int *d_zsrc; hipMalloc(&d_zsrc,nshot*sizeof(int));
    int *d_xrec; hipMalloc(&d_xrec,nrecs*sizeof(int)); 
    int *d_yrec; hipMalloc(&d_yrec,nrecs*sizeof(int)); 
    int *d_zrec; hipMalloc(&d_zrec,nrecs*sizeof(int)); 

    float *d_seismPs; hipMalloc(&d_seismPs,nt*nrecs*sizeof(float));
    float *d_seismVx; hipMalloc(&d_seismVx,nt*nrecs*sizeof(float));
    float *d_seismVy; hipMalloc(&d_seismVy,nt*nrecs*sizeof(float));
    float *d_seismVz; hipMalloc(&d_seismVz,nt*nrecs*sizeof(float));
    
    float *d_seismP;   hipMalloc(&d_seismP,nt*nrecs*sizeof(float));
    float *d_seismSv;  hipMalloc(&d_seismSv,nt*nrecs*sizeof(float));
    float *d_seismShx; hipMalloc(&d_seismShx,nt*nrecs*sizeof(float));
    float *d_seismShy; hipMalloc(&d_seismShy,nt*nrecs*sizeof(float));

    /* Host inputs and calculations */

    importFloatVector(vp,nPoints,argv[2]);
    importFloatVector(vs,nPoints,argv[3]);
    importFloatVector(rho,nPoints,argv[4]);
    importFloatVector(damp,nPoints,argv[5]);

    importFloatVector(source,nsrc,argv[6]);
    importIntegerVector(xsrc,nshot,argv[7]);
    importIntegerVector(ysrc,nshot,argv[8]);
    importIntegerVector(xrec,nrecs,argv[9]);
    importIntegerVector(yrec,nrecs,argv[10]);

    for (int index = 0; index < nPoints; index++)
    {
        M[index] = rho[index]*pow(vs[index],2.0f);
        L[index] = rho[index]*pow(vp[index],2.0f) - 2.0f*M[index];
    }

    for (int i = 0; i < nshot; i++) zsrc[i] = nabc + 5;
    for (int i = 0; i < nrecs; i++) zrec[i] = nabc + wbh;
    
    /* Managing memory */
    
    hipMemcpy(d_rho,rho,nPoints*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_M,M,nPoints*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_L,L,nPoints*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_damp,damp,nPoints*sizeof(float),hipMemcpyHostToDevice);

    hipMemcpy(d_source,source,nsrc*sizeof(float),hipMemcpyHostToDevice);

    hipMemcpy(d_xsrc,xsrc,nshot*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_ysrc,ysrc,nshot*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_zsrc,zsrc,nshot*sizeof(int),hipMemcpyHostToDevice);
    
    hipMemcpy(d_xrec,xrec,nrecs*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_yrec,yrec,nrecs*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_zrec,zrec,nrecs*sizeof(int),hipMemcpyHostToDevice);

    printf("Wave propagation in elastic isotropic 3D media\n\n");    
    for (int shotPointer = floor(nshot/2 - nrecx/2); shotPointer < floor(nshot/2 - nrecx/2 + 1); shotPointer++)
    {
        setWaveField<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_Txx,d_Tyy,d_Tzz,d_Txz,d_Tyz,d_Txy,nPoints);

        for (int timePointer = 0; timePointer < nt; timePointer++) 
        {
            if (timePointer % (nt/10) == 0) printf("Propagation time = %.3f\n",timePointer*dt);

            computeStress<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_Txx,d_Tyy,d_Tzz,d_Txy,d_Txz,d_Tyz,d_M,d_L,d_source,d_xsrc,d_ysrc,d_zsrc,nsrc,timePointer,shotPointer,nxx,nyy,nzz,dx,dy,dz,dt);        
            hipDeviceSynchronize();

            computeVelocity<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_Txx,d_Tyy,d_Tzz,d_Txy,d_Txz,d_Tyz,d_rho,nxx,nyy,nzz,dx,dy,dz,dt);    
            hipDeviceSynchronize();

            cerjanElasticABC3D<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_Txx,d_Tyy,d_Tzz,d_Txy,d_Txz,d_Tyz,d_damp,nPoints);
            hipDeviceSynchronize();    

            getPressureWaveField<<<nPoints/threads,threads>>>(d_Txx,d_Tyy,d_Tzz,d_Ps,nPoints);
            hipDeviceSynchronize();    
            
            getPWaveField<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_P,nxx,nyy,nzz,dx,dy,dz);
            hipDeviceSynchronize();    
          
            getSWaveField<<<nPoints/threads,threads>>>(d_Vx,d_Vy,d_Vz,d_Shx,d_Shy,d_Sv,nxx,nyy,nzz,dx,dy,dz);
            hipDeviceSynchronize();    

            getSeismogram<<<1,nrecs>>>(d_seismVx,d_Vx,d_xrec,d_yrec,d_zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer);
            hipDeviceSynchronize();    

    //         getSeismogram(seismVy,Vy,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismVz,Vz,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismPs,Ps,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);

    //         getSeismogram(seismP,P,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismSv,Sv,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismShx,Shx,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
    //         getSeismogram(seismShy,Shy,xrec,yrec,zrec,nrecx,nrecy,nrecs,nt,nxx,nyy,nzz,timePointer,nsrc,dt);
        }
    }
    
    hipMemcpy(seismVx,d_seismVx,nt*nrecs*sizeof(float),hipMemcpyDeviceToHost);

    exportVector(seismVx,nt*nrecs,(char *)"results/seismVx.bin");
    // exportVector(seismVy,nt*nrecs,(char *)"results/seismVy.bin");
    // exportVector(seismVz,nt*nrecs,(char *)"results/seismVz.bin");
    // exportVector(seismPs,nt*nrecs,(char *)"results/seismPs.bin");

    // exportVector(seismP,nt*nrecs,(char *)"results/seismP.bin");
    // exportVector(seismSv,nt*nrecs,(char *)"results/seismSv.bin");
    // exportVector(seismShx,nt*nrecs,(char *)"results/seismShx.bin");
    // exportVector(seismShy,nt*nrecs,(char *)"results/seismShy.bin");

    t_f = time(NULL);
    total_time = difftime(t_f, t_0);
    printf("\nExecution time: \033[31m%.0fs\n\033[m", total_time);    
    return 0;
} 